
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU()
{
  printf("Hello World from GPU! thread\n");
}

int main(int argc, char ** argv)
{
  printf("Hello World from CPU!\n");
  dim3 block(10,1);
  dim3 grid(1,1);
  helloFromGPU <<<grid,block>>>();
  hipDeviceReset();
  return 0;
}
