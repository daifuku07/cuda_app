
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
#include <time.h>

const int N = 1024; // 正方行列のサイズを指定（N×N）
const int BLOCK = 16; // ブロックのサイズを指定

double cpuSecond();

__global__ void matrixMul(int *dMatA, int *dMatB, int *dMatC)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int scan;
  int target = 0;

  // 行列の演算を行う
}

int main(int argc, char** argv)
{
    // 行列のサイズをバイト単位で算出
    int matrixSize = sizeof(unsigned int) * N * N;

    int test1, test2;

    double start;
    double gpucalctime;
    double cpucalctime;

    // ホスト側の行列変数設定
    int* hMatA;
    int* hMatB;
    int* hMatC;

    // 行列変数のメモリ確保
    hMatA = (int*)malloc(matrixSize);
    hMatB = (int*)malloc(matrixSize);
    hMatC = (int*)malloc(matrixSize);


    // 行列の初期値設定
    // mat[row][col] を一次元配列として格納
    int row, col, scan;
    srand((unsigned)time(NULL));
    for (row = 0; row < N; row++){
        for (col = 0; col < N; col++){
            hMatA[row * N + col] = rand() % (N * N);
            hMatB[row * N + col] = rand() % (N * N);
            hMatC[row * N + col] = 0;
        }
    }
    /* CPU側での処理時間計測 */
    start = cpuSecond();

    for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            for (scan = 0; scan < N; scan++) {
                hMatC[row * N + col] += hMatA[row * N + scan] * hMatB[scan * N + col];
            }
        }
    }

    cpucalctime = cpuSecond() - start;


    test1 = hMatC[52];

    /* GPU側での処理時間計測 */
    // デバイス側の行列変数設定
    int* dMatA;
    int* dMatB;
    int* dMatC;

    // デバイスメモリ領域の確保
    hipMalloc((void**)&dMatA, matrixSize);
    hipMalloc((void**)&dMatB, matrixSize);
    hipMalloc((void**)&dMatC, matrixSize);

    // GPU 乗算及び時間計測
    start = cpuSecond();

    // ホストからデバイスへの変数の受け渡し
    hipMemcpy(dMatA, hMatA, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(dMatB, hMatB, matrixSize, hipMemcpyHostToDevice);

    // ブロックサイズとグリッドサイズの設定
    dim3 block(BLOCK, BLOCK);
    dim3 grid( N / BLOCK, N / BLOCK);

    // カーネルの起動
    matrixMul<<<grid, block>>>(dMatA, dMatB, dMatC);
    hipDeviceSynchronize();

    // 結果の領域確保とデバイス側からのメモリ転送
    hipMemcpy(hMatC, dMatC, matrixSize, hipMemcpyDeviceToHost);

    gpucalctime = cpuSecond() - start;

    test2 = hMatC[52];


    // 結果の出力
    printf("[CPU]calc exetime : %f s.\n", cpucalctime);
    printf("[GPU]calc exetime : %f s.\n", gpucalctime);
    printf("GPUはCPUの処理を　%f 倍高速化 \n",cpucalctime/gpucalctime);

    printf("配列52をみて演算結果の確認\n");
    printf("cpu %d , gpu %d \n", test1, test2);

    // ホスト・デバイスメモリの解放
    free(hMatA);
    free(hMatB);
    free(hMatC);
    hipFree(dMatA);
    hipFree(dMatB);
    hipFree(dMatC);

    // 終了処理
    hipDeviceReset();
    return 0;
}

/* 時間を秒で返す*/
double cpuSecond()
{
    struct timespec tp;
    clock_gettime(CLOCK_REALTIME, &tp);
    return((double)tp.tv_sec + (double)tp.tv_nsec * 1.e-9);
}
